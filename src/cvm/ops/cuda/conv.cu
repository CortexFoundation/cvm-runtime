#include "hip/hip_runtime.h"
#include "cuda_ops.h"
#include "../common.h"
#include <assert.h>

namespace cvm{
namespace runtime{

#define BS 16
#define FS 8

  __global__ void kernel_int32_to_int8(const int32_t *in_data, int8_t *out_data, const int n){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for(int64_t i = tid; i < n; i+= gridDim.x * blockDim.x){
      out_data[i] = static_cast<int8_t>(in_data[i]);
    }
  }

__global__ void kernel_transpose_i32_to_i8(const int32_t *in, int8_t *out, 
    const int32_t H, const int32_t W, 
    const int32_t OH, const int32_t OW){
  int bidy = blockIdx.y;
  int bidx = blockIdx.x; 
  int lidy = threadIdx.y;
  int lidx = threadIdx.x;
  __shared__ int32_t share_in[32][33];
  int y = bidy * blockDim.y + lidy;
  int x = bidx * blockDim.x + lidx;
  if(y < H && x < W){
    share_in[lidx][lidy] = in[y * W + x];
  }
  __syncthreads();
  int oy = bidx * blockDim.x + lidy;
  int ox = bidy * blockDim.y + lidx;
  if(oy < W && ox < H)
    out[oy * OH + ox] = (int8_t)share_in[lidy][lidx];
}

__global__ void im2col_gpu_kernel_pad(const int n, const int32_t* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
    int8_t* data_col) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const int cols = height_col * width_col;
  const int offset = (cols + 63) / 64 * 64;
  for(int64_t index = tid; index < n; index += gridDim.x*blockDim.x){
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    int8_t* data_col_ptr = data_col;
    data_col_ptr += c_col * offset + h_col * width_col + w_col;//(c_col * height_col + h_col) * width_col + w_col;
    const int32_t* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        *data_col_ptr =
          (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
          static_cast<int8_t>(data_im_ptr[i * dilation_h * width + j * dilation_w]) : 0;
        data_col_ptr += offset;
      }
    }
  }
}
__global__ void im2col_gpu_kernel(const int n, const int32_t* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
    int8_t* data_col) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(int64_t index = tid; index < n; index += gridDim.x*blockDim.x){
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    int8_t* data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const int32_t* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        *data_col_ptr =
          (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
          static_cast<int8_t>(data_im_ptr[i * dilation_h * width + j * dilation_w]) : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

#define TILE_WIDTH 16
template<const bool has_bias, int NUM>
__global__ void kernel_matrix_mul(
    int8_t *a, // m*k 
    int8_t *b, // k*n
    int32_t *c, // m*n
    int32_t m, int32_t k, int32_t n, int32_t *bias){
  __shared__ int8_t sharedm[TILE_WIDTH*NUM][TILE_WIDTH*NUM];
  __shared__ int8_t sharedn[TILE_WIDTH*NUM][TILE_WIDTH*NUM];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by*TILE_WIDTH*NUM + ty;
  int col = bx*TILE_WIDTH*NUM + tx;
  int sum[NUM][NUM]= {{0}};

  for (int i = 0; i < (int)(ceil((float)k/TILE_WIDTH)); i+=NUM)
  {
    for(int ii = 0; ii < NUM; ++ii){
      int r_offset = ii * TILE_WIDTH;
      for(int jj = 0; jj < NUM; ++jj){
        int c_offset = jj * TILE_WIDTH;
        int arow_offset = row + r_offset;
        int acol_offset = i*TILE_WIDTH + tx + c_offset;
        int brow_offset = i*TILE_WIDTH + ty + r_offset;
        int bcol_offset =  col + c_offset;

        if(arow_offset < m && acol_offset < k)
          sharedm[ty+r_offset][tx+c_offset] = a[(arow_offset)*k + acol_offset];
        else sharedm[ty+r_offset][tx+c_offset] = 0;

        if(brow_offset < k && bcol_offset < n)
          sharedn[ty+r_offset][tx+c_offset] = b[(brow_offset)*n + bcol_offset];
        else sharedn[ty+r_offset][tx+c_offset] = 0;
      }
    }
    __syncthreads();

    for(int j = 0; j < TILE_WIDTH; j++){ 
      int8_t tm[NUM][NUM], tn[NUM][NUM];
#pragma unroll
      for(int ii = 0; ii < NUM; ++ii){
#pragma unroll
        for(int jj = 0; jj < NUM; ++jj){
          tm[ii][jj] = sharedm[ty+ii*TILE_WIDTH][j+jj*TILE_WIDTH];
          tn[ii][jj] = sharedn[j+ii*TILE_WIDTH][tx+jj*TILE_WIDTH];
        }
      }
      for(int ii = 0; ii < NUM; ++ii){
#pragma unroll
        for(int kk = 0; kk < NUM; ++kk){
#pragma unroll
          for(int jj = 0; jj < NUM; ++jj){
            sum[ii][jj] += tm[ii][kk] * tn[kk][jj];
          }
        }
      }
    }
    __syncthreads();
  }
  if(has_bias) {
    for(int ii = 0; ii < NUM; ++ii){
      int32_t bv = bias[row + ii * TILE_WIDTH];
      for(int jj = 0; jj < NUM ;++jj){
        sum[ii][jj] += bv; 
      }
    }
  }
  for(int ii = 0; ii < NUM; ++ii){
    int c_r_offset = row + ii * TILE_WIDTH;
    for(int jj = 0; jj < NUM ;++jj){
      int c_c_offset = col + jj * TILE_WIDTH;
      if(c_r_offset < m && c_c_offset < n)
        c[(c_r_offset)*n + c_c_offset] = sum[ii][jj];
    }
  }
}

template<bool has_bias>
__global__ void kernel_matrix_mul_opt(
    char4 *A, // k*m 
    char4  *B, // k*n
    int32_t *C, // m*n
    int32_t M, int32_t K, int32_t N, int32_t *bias,
    const int32_t TM, const int32_t TN, const int32_t TK){
  int lidx = threadIdx.x;
  int lidy = threadIdx.y;
  int bidx = blockIdx.x;
  int bidy = blockIdx.y;

  int aBegin = bidy * TILE_WIDTH;
  int aStep = TILE_WIDTH * (TM/4);
  int bBegin = bidx * TILE_WIDTH;
  int bStep = TILE_WIDTH * (TN/4);

  int round_K = TK / TILE_WIDTH;
  int32_t csub[4][4] = {{0}};
  for(int i = 0, a = aBegin, b = bBegin; i < round_K; ++i, a += aStep, b+= bStep){
    __shared__ char4 share_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ char4 share_b[TILE_WIDTH][TILE_WIDTH];

    //int aid = a + lidy * (TM/4) + lidx;
    share_a[lidy][lidx] = A[a + lidy * (TM/4) + lidx];

    //int bid = b + lidy * (TN/4) + lidx;
    share_b[lidy][lidx] = B[b + lidy * (TN/4) + lidx];
    __syncthreads();

    for(int k = 0; k < TILE_WIDTH; ++k){
      signed char pa[4] = {share_a[k][lidy].x, share_a[k][lidy].y, share_a[k][lidy].z, share_a[k][lidy].w};
      signed char pb[4] = {share_b[k][lidx].x, share_b[k][lidx].y, share_b[k][lidx].z, share_b[k][lidx].w};
#pragma unroll
      for(int ii = 0; ii < 4; ii++){
#pragma unroll
        for(int jj = 0; jj < 4; jj++){
          csub[ii][jj] += pa[ii] * pb[jj];
        }
      }
    }
    __syncthreads();
  }

 // int c = bidy * TILE_WIDTH * N + bidx * TILE_WIDTH;
  int gidy = bidy * TILE_WIDTH + lidy;
  int gidx = bidx * TILE_WIDTH + lidx;
  for(int ii = 0; ii < 4; ii++){
    int row = (gidy * 4 + ii);
    int bv = 0;
    if(has_bias && row < M){
      bv = bias[row]; 
    }
    for(int jj = 0; jj < 4; jj++){
      int col = gidx * 4 + jj;
      if(row < M && col < N)
      C[row * N + col] = csub[ii][jj] + bv;
    }
  }
}

inline void im2col_gpu(const int32_t* data_im, const int channels,
        const int height, const int width, const int kernel_h, const int kernel_w,
        const int pad_h, const int pad_w,
        const int stride_h, const int stride_w,
        const int dilation_h, const int dilation_w,
        int8_t* data_col) {
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad_h -
            (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    int width_col = (width + 2 * pad_w -
            (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    int num_kernels = channels* height_col* width_col;
    int threads = 256;
    int blocks = (num_kernels + threads - 1) / threads;
    im2col_gpu_kernel_pad<<<blocks, threads>>>(
                num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
                pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
                width_col, data_col);
}

const char* cuda_conv2d(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, const int32_t f_h, const int32_t f_w,
    int32_t *bias,
    const int32_t padding_h, const int32_t padding_w,
    const int32_t stride_h, const int32_t stride_w,
    const int32_t dilation_h, const int32_t dilation_w,
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, 
    int32_t device_id,
    int32_t *ext_space, 
    int32_t ext_space_size, int& error_code){

  if(i_n < 1 || i_c < 1 || i_h < 1 || i_w < 1 || f_n < 1 || f_c < 1 || f_h < 1 || f_w < 1 || 
      padding_h < 0 || padding_w < 0 || stride_h < 1 || stride_w < 1 || dilation_h < 1 || dilation_w < 1 ||
      o_n < 1 || o_c < 1 || o_h < 1 || o_w < 1){
    error_code = ERROR_PARAMS;
    return "error args";
  }
  int32_t *dev_i = input, *dev_f = filter, *dev_o = output, *dev_b = bias;

  const int M = o_c;
  const int TM = (M + 63) / 64* 64;
  const int K = i_c * f_h * f_w;
  const int TK = (K + 63) / 64 * 64;
  const int N = o_h * o_w;
  const int TN = (N + 63) / 64 * 64;
  dim3 bDim(TILE_WIDTH, TILE_WIDTH, 1);
  int gh = (TM/4 + TILE_WIDTH - 1) / TILE_WIDTH;
  int gw = (TN/4 + TILE_WIDTH - 1) / TILE_WIDTH;
  dim3 gDim(gw, gh, 1);

  hipMemset(ext_space, 0, sizeof(int32_t)*ext_space_size);
  int8_t *d_f = (int8_t*)ext_space;
  int8_t *d_col = d_f + TM * TK;

 // int blockSize = 256;
 // int gridSize = getGridSize(fn, blockSize);
  //kernel_int32_to_int8<<<gridSize, blockSize>>>(dev_f, d_f, fn);
  dim3 bSize(32, 32, 1);
  dim3 gSize((K+31)/32, (M+31)/32, 1);
  kernel_transpose_i32_to_i8<<<gSize, bSize>>>(dev_f, d_f, M, K, TM, TK);

  for(int i = 0; i < o_n; i++){
    im2col_gpu(dev_i + i * i_c * i_h * i_w,
        i_c, i_h, i_w, f_h, f_w, padding_h, padding_w, stride_h, stride_w, 
        dilation_h, dilation_w, d_col);
    if(dev_b == NULL)
      kernel_matrix_mul_opt<false><<<gDim, bDim>>>((char4*)d_f, (char4*)d_col, dev_o + i * o_c * o_h * o_w, M, K, N, dev_b, TM, TN, TK);
    else
      kernel_matrix_mul_opt<true><<<gDim, bDim>>>((char4*)d_f, (char4*)d_col, dev_o + i * o_c * o_h * o_w, M, K, N, dev_b, TM, TN, TK);
  }

  print_to_file(dev_i, o_n * i_c* i_h * i_w, "conv2d_x.txt");
  print_to_file(dev_o, o_n * o_c * o_h * o_w, "conv2d.txt");
  //return check_cuda_error(error);
  return "";
}
__global__ void kernel_depthwise_conv2d(
    const int32_t * __restrict__ input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    const int32_t * __restrict__ filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    const int32_t * __restrict__ bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w, 
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w)
{
  int g_x = blockDim.x * blockIdx.x + threadIdx.x;
  int l_y = threadIdx.y; 
  int l_x = threadIdx.x;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; // for stride
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int perBlockOneImageY = (tmp_o_h+BS-1) / BS;
  int perBlockOneImageX = (tmp_o_w+BS-1) / BS;
  int l_o_c = blockIdx.y / perBlockOneImageY;
  int l_f_c = l_o_c % o_c;
  int l_o_hi = blockIdx.y % perBlockOneImageY;
  int l_o_wi = blockIdx.x % perBlockOneImageX;
  int l_o_h = l_o_hi * BS + l_y;
  //    int l_o_w = l_o_wi * BS + l_x;
  if(l_o_h >= tmp_o_h || g_x >= tmp_o_w) return;

  const int32_t F_H = f_h;
  const int32_t F_W = f_w;
  //    __shared__ int32_t shared_i[BS + F_H - 1][BS + F_W - 1];
  int32_t sih = BS + tmp_f_h - 1;
  int32_t siw = BS + tmp_f_w - 1;
  extern __shared__ int32_t  share[];
  int32_t *shared_i = (int32_t*)share; 
  int32_t *shared_f = &share[sih * siw];

  int32_t sum = 0; 
  int min_s_y = (l_o_hi+1) * BS <= tmp_o_h ? BS : tmp_o_h%BS;
  int min_s_x = (l_o_wi+1) * BS <= tmp_o_w ? BS : tmp_o_w%BS;

  //load input to shared
  int l_i_h = l_o_h - padding_h;
  int i_y = l_o_c * i_h + l_i_h;
  int i_x = g_x - padding_w;
  // 0~2-> -1~1
  if(l_i_h < 0 || i_x < 0 || l_i_h >= i_h || i_x >= i_w)
    shared_i[l_y*siw + l_x] = 0;
  else
    shared_i[l_y*siw + l_x] = input[i_y * i_w + i_x];

  if(l_y < tmp_f_h-1){
    for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
      if(l_i_h+min_s_y+i-l_y < 0 || i_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x >= i_w)
        shared_i[(i+min_s_y)*siw + l_x] = 0;
      else
        shared_i[(i + min_s_y)*siw + l_x] = input[(i_y + min_s_y + i - l_y) * i_w + i_x]; 
    }
  }
  if(l_x < tmp_f_w-1){
    for(int i = l_x; i < tmp_f_w-1; i+= min_s_x){
      if(l_i_h < 0 || i_x+min_s_x+i-l_x < 0 || l_i_h >= i_h || i_x+min_s_x+i-l_x >= i_w)
        shared_i[l_y * siw + i+min_s_x] = 0;
      else
        shared_i[l_y * siw + i + min_s_x] = input[i_y * i_w + i_x + min_s_x + i - l_x];
    }
  }
  if(l_y < tmp_f_h-1 && l_x < tmp_f_w-1){
    for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
      for(int j = l_x; j < tmp_f_w-1; j+=min_s_x){
        if(l_i_h+min_s_y+i-l_y < 0 || i_x+min_s_x+j-l_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x+min_s_x+j-l_x >= i_w)
          shared_i[(i+min_s_y) * siw + j+min_s_x] = 0;
        else
          shared_i[(i+min_s_y) * siw + j+min_s_x] = input[(i_y+min_s_y + i-l_y)*i_w + i_x + min_s_x + j - l_x];
      }
    }
  }

  //load filter to shared;
  if(l_y < F_H && l_x < F_W){
    for(int i = l_y; i < F_H; i+= min_s_y)
      for(int j = l_x; j < F_W; j+=min_s_x)
        shared_f[i*F_W + j] = filter[l_f_c * F_H * F_W + i * F_W + j];
  }
  __syncthreads();

  for(int fy = 0; fy < F_H; fy++){
    for(int fx = 0; fx < F_W; fx++){
      sum += shared_i[(l_y+fy*dilation_h)*siw + l_x+fx*dilation_w] * shared_f[fy*F_W + fx];
    }
  } 
  __syncthreads();

  if(l_o_h % stride_h == 0 && g_x % stride_w == 0){
    //int oi = l_o_c * o_h * o_w + l_o_h * o_w + g_x;
    int oi = l_o_c * o_h * o_w + l_o_h/stride_h * o_w + g_x/stride_w;
    output[oi] = sum + (bias != NULL ? bias[l_o_c%o_c] : 0);
  }
}
__global__ void kernel_depthwise_conv2d_no_shared(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    int32_t *bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w, 
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t sum = 0;
    for(int fy = 0; fy < f_h; ++fy){
      for(int fx = 0; fx < f_w; ++fx){
        int32_t l_i_h = l_o_h * stride_h + fy * dilation_h - padding_h;
        int32_t l_i_w = gx * stride_w + fx * dilation_w - padding_w;
        int32_t x;
        if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
          //x = 0;
          continue;
        x = input[l_o_n * i_c * i_h * i_w + l_o_c * i_h * i_w + l_i_h * i_w + l_i_w];
        sum += x * filter[l_o_c * f_h * f_w + fy * f_w + fx];
      }
    }
    output[gy * o_w + gx] = sum + (bias != NULL ? bias[l_o_c] : 0);
  }
}
const char* cuda_depthwise_conv2d(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    int32_t *bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w,
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, int32_t device_id, int& error_code){
  int32_t *dev_i = input, *dev_f = filter, *dev_o = output, *dev_b = bias;

  int b_h = BS;
  int b_w = BS;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; //for stride > 1
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int32_t g_h = o_n * o_c * ((tmp_o_h + b_h - 1) / b_h); 
  int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
  dim3 bDim(b_w, b_h, 1);
  dim3 gDim(g_w, g_h, 1);
  kernel_depthwise_conv2d_no_shared<<<gDim, bDim>>>(
      dev_i, i_n, i_c, i_h, i_w,
      dev_f, f_n, f_c, f_h, f_w,
      dev_b, 
      padding_h, padding_w,
      stride_h, stride_w,
      dilation_h, dilation_w,
      groups,
      dev_o, o_n, o_c, o_h, o_w);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}
__global__ void kernel_groupwise_conv2d_no_shared(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    int32_t *bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w, 
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  const int32_t ochannels_per_group = o_c / groups;
  const int32_t ichannels_per_group = i_c / groups;
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t sum = 0;
    int32_t ic = l_o_c / ochannels_per_group * ichannels_per_group;
    for(int tic = 0; tic < ichannels_per_group; ++tic){
      for(int fy = 0; fy < f_h; ++fy){
        for(int fx = 0; fx < f_w; ++fx){
          int32_t l_i_h = l_o_h * stride_h + fy * dilation_h - padding_h;
          int32_t l_i_w = gx * stride_w + fx * dilation_w - padding_w;
          int32_t x;
          if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
            continue;
          x = input[l_o_n * i_c * i_h * i_w + (ic+tic) * i_h * i_w + l_i_h * i_w + l_i_w];
          sum += x * filter[l_o_c * f_h * f_w * f_c + tic * f_h * f_w + fy * f_w + fx];
        }
      }
    }
    output[gy * o_w + gx] = sum + (bias != NULL ? bias[l_o_c] : 0);
  }
}
const char* cuda_groupwise_conv2d(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    int32_t *bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w,
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, int32_t device_id, int& error_code){
  int32_t *dev_i = input, *dev_f = filter, *dev_o = output, *dev_b = bias;

  int b_h = BS;
  int b_w = BS;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; 
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; 
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int32_t g_h = o_n * o_c * ((tmp_o_h + b_h - 1) / b_h); 
  int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
  dim3 bDim(b_w, b_h, 1);
  dim3 gDim(g_w, g_h, 1);
  kernel_groupwise_conv2d_no_shared<<<gDim, bDim>>>(
      dev_i, i_n, i_c, i_h, i_w,
      dev_f, f_n, f_c, f_h, f_w,
      dev_b, 
      padding_h, padding_w,
      stride_h, stride_w,
      dilation_h, dilation_w,
      groups,
      dev_o, o_n, o_c, o_h, o_w);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}

__global__ void kernel_max_pool_no_shared(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t f_h, int32_t f_w,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t minV = (int32_t)1 << 31;
    int32_t maxV = minV;
    for(int fy = 0; fy < f_h; ++fy){
      for(int fx = 0; fx < f_w; ++fx){
        int32_t l_i_h = l_o_h * stride_h + fy  - padding_h;
        int32_t l_i_w = gx * stride_w + fx - padding_w;
        int32_t x;
        if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
          x = minV;
        else x = input[l_o_n * i_c * i_h * i_w + l_o_c * i_h * i_w + l_i_h * i_w + l_i_w];
        maxV = maxV < x ? x : maxV;
      }
    }
    output[gy * o_w + gx] = maxV;
  }
}
const char* cuda_max_pool(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    const int32_t f_h, const int32_t f_w,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, int32_t device_id, int& error_code){
  int32_t *dev_i = input, *dev_o = output;

  int b_h = BS;
  int b_w = BS;
  int32_t g_h = o_n * o_c * ((o_h + b_h - 1) / b_h); 
  int32_t g_w = (o_w + b_w - 1) / b_w;
  dim3 bDim(b_w, b_h, 1);
  dim3 gDim(g_w, g_h, 1);
  kernel_max_pool_no_shared<<<gDim, bDim>>>(
      dev_i, i_n, i_c, i_h, i_w,
      f_h, f_w,
      padding_h, padding_w, 
      stride_h, stride_w,
      dev_o, o_n, o_c, o_h, o_w);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}
}
}
